#include "hip/hip_runtime.h"
#include "common.h"
#include "hip/hip_runtime.h" 
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

struct point{
    double x;
    double y;
};

__global__ void testFunc(point* d_a)
{
    if(threadIdx.x == 0 && threadIdx.y == 0)
    {  
       d_a->x=100.0;  
       d_a->y = 100.0;    
    }
}

int main()
{ 
    point *cpu_pt, *gpu_pt;
    cpu_pt = (point*)malloc(sizeof(point));
    cpu_pt->x = 10;
    cpu_pt->y = 10;
    hipMalloc((void**)&gpu_pt, sizeof(point));
    hipMemcpy(gpu_pt, cpu_pt, sizeof(point), hipMemcpyHostToDevice);
    dim3 dimblock(16, 16);
    dim3 dimgrid(1, 1);
    testFunc<<<dimgrid, dimblock>>>(gpu_pt);
    hipMemcpy(cpu_pt, gpu_pt,sizeof(point),hipMemcpyDeviceToHost); 
    printf("cpu_pt->x is %lf, cpu_pt->y is %lf\n", cpu_pt->x,cpu_pt->y);
    return 0;

}